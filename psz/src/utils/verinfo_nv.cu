#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* This sample queries the properties of the CUDA devices present in the system
 * via CUDA Runtime API. */

// modified from deviceQuery.cpp of CUDA Samples

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "utils/verinfo.h"

void print_NVCC_ver()
{
  printf(
      "- nvcc: %s.%s.%s\n",  //
      STRINGIZE_VALUE_OF(__CUDACC_VER_MAJOR__),
      STRINGIZE_VALUE_OF(__CUDACC_VER_MINOR__),
      STRINGIZE_VALUE_OF(__CUDACC_VER_BUILD__));
}

int print_CUDA_driver()
{
  hipError_t result;
  int driver_version = 0;

  std::cout << "- CUDA driver: ";

  result = hipInit(0);
  if (result != hipSuccess) {
    const char* error_str;
    hipDrvGetErrorString(result, &error_str);
    std::cerr << "(failed to initialize: " << error_str << ")" << std::endl;
    return 1;
  }

  // Get the CUDA driver version
  result = hipDriverGetVersion(&driver_version);
  if (result != hipSuccess) {
    const char* error_str;
    hipDrvGetErrorString(result, &error_str);
    std::cerr << "(failed to get driver version: " << error_str << ")"
              << std::endl;
    return 1;
  }

  int major_version = driver_version / 1000;
  int minor_version = (driver_version % 1000) / 10;
  std::cout << major_version << "." << minor_version << std::endl;

  return 0;
}

template <class T>
inline void getCudaAttribute(
    T* attribute, hipDeviceAttribute_t device_attribute, int device)
{
  hipDeviceGetAttribute(attribute, device_attribute, device);
}

void CUDA_devices()
{
  int device_count = 0;
  hipGetDeviceCount(&device_count);

  if (device_count == 0) { printf("0 devices detected\n"); }
  else {
    printf("%d CUDA device(s):\n", device_count);
  }

  for (auto dev = 0; dev < device_count; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    auto membw_GiBps = membw_base1024(
        deviceProp.memoryBusWidth, deviceProp.memoryClockRate * 1e3);
    auto membw_GBps = membw_base1000(
        deviceProp.memoryBusWidth, deviceProp.memoryClockRate * 1e3);

    printf("- %s\n", deviceProp.name);
    printf(
        "  - %d multiprocessors; SM capability: %d.%d; warp size: %d\n",
        deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor,
        deviceProp.warpSize);
    printf(
        "  - global VRAM: %.0f MB (theoretically) at \n",
        (float)deviceProp.totalGlobalMem / 1048576.0f);
    printf(
        "    %.1f GiB/s (base-1024) or %.1f GB/s (base-1000)\n", membw_GiBps,
        membw_GBps);
    printf("  - L2 cache: %d bytes\n", deviceProp.l2CacheSize);
    printf(
        "  - per-block/SM total shared memory: %zu/%zu bytes\n",
        deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerMultiprocessor);
    // printf("  - total constant memory: %zu bytes\n",
    // deviceProp.totalConstMem);
    printf(
        "  - per-block/SM max thread count: %d/%d\n",
        deviceProp.maxThreadsPerBlock, deviceProp.maxThreadsPerMultiProcessor);
    printf(
        "  - max thread-block dim (x,y,z): (%d, %d, %d)\n",
        deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
        deviceProp.maxThreadsDim[2]);
    printf(
        "  - per-block total registers count: %d\n", deviceProp.regsPerBlock);
  }
}